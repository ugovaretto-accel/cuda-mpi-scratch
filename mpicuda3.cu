#include "hip/hip_runtime.h"
// "mpi + cuda reduction + timing" 

#ifdef GPU
#include <hip/hip_runtime.h>
#endif
#include <mpi.h>
#include <iostream>
#include <vector>
#include "mpierr.h"
#include <cmath>
#include <algorithm>
#include <sstream>
#include <string>
#include <set>
#include <numeric>
#include <ctime>

// switches:
// #GPU : enable GPU computation
// #NO_LOG: do not printout log messages
// #REDUCE_CPU: perform final per-task reduction step on the CPU
// #DOUBLE_: double precision
// #MPI_RROBIN_: assume a round robin layout i.e process 0 -> node 0, process 1 -> node 1 ...
// #NO_GPU_MALLOC_TIME: do not take into account malloc time; usually this is part of an initialization step


// compilation with mvapich2:
// nvcc -L/apps/eiger/mvapich2/1.6/mvapich2-gnu/lib -I/apps/eiger/mvapich2/1.6/mvapich2-gnu/include \
// -libumad -lmpich -lpthread -lrdmacm -libverbs -arch=sm_20 -DGPU \
// ~/projects/gpu-training/trunk/cuda_exercises_ugo/resources/mpiscratch/mpicuda2.cu


// run:
// 1) w/o scheduler: mpiexec -np ... -hosts ... ./a.out
// 2) w/ scheduler: see mpi_cuda_pbs_ref.sh script

// note: when using mvapich2/1.6 and *not* going through the pbs scheduler it seems
//       the default behavior is rrobin, using the pbs launch script the default
//       behavior is "bunch" (as defined by the mvapich2 documentation) 

// note: using single precision floats because that's the only supported type
//       for atomics on CUDA 4

// note: experiment with different number of MPI tasks per GPU/node; using
//       256 Mi floats, 16 MPI tasks on two nodes (8 per node, 4 per GPUs)
//       CUDA fails to allocate memory exaclty for one task on each node;
//       Everything works fine with the same data with 8 tasks (4 per node, 2 per GPU ).

// note: it is possible to implement a discovery step to find the current MPI layout
//       by checking if MPI rank 0 and 1 are on the same processor ("bunch" layout) or
//       not ("scatter" layout)


#ifndef DOUBLE_
// with CUDA 4.0 atomics are available for single precision only!!!
typedef float real_t;
#define MPI_REAL_T_ MPI_FLOAT
#else
typedef double real_t;
#define MPI_REAL_T_ MPI_DOUBLE
#endif

//------------------------------------------------------------------------------
#ifdef GPU
const int BLOCK_SIZE = 512;
#ifndef DOUBLE_ //atomics are available for single precision only!!!
__global__ void dot_product_kernel( const real_t* v1, const real_t* v2, int N, real_t* out ) {
    __shared__ real_t cache[ BLOCK_SIZE ];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if( i >= N ) return;
    cache[ threadIdx.x ] = 0.f;
    while( i < N ) {
        cache[ threadIdx.x ] += v1[ i ] * v2[ i ];
        i += gridDim.x * blockDim.x;
    }    
    i = BLOCK_SIZE / 2;
    while( i > 0 ) {
        if( threadIdx.x < i ) cache[ threadIdx.x ] += cache[ threadIdx.x + i ];
        __syncthreads();
        i /= 2; //not sure bitwise operations are actually faster
    }
    if( threadIdx.x == 0 ) atomicAdd( out, cache[ 0 ] );   
}
#endif

__global__ void partial_dot_product_kernel( const real_t* v1, const real_t* v2, int N, real_t* out ) {
    __shared__ real_t cache[ BLOCK_SIZE ];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if( i >= N ) return;
    cache[ threadIdx.x ] = 0.f;
    while( i < N ) {
        cache[ threadIdx.x ] += v1[ i ] * v2[ i ];
        i += gridDim.x * blockDim.x;
    }    
    i = BLOCK_SIZE / 2;
    while( i > 0 ) {
        if( threadIdx.x < i ) cache[ threadIdx.x ] += cache[ threadIdx.x + i ];
        __syncthreads();
        i /= 2; //not sure bitwise operations are actually faster
    }
    if( threadIdx.x == 0 ) out[ blockIdx.x ] = cache[ 0 ];
}
#endif


//------------------------------------------------------------------------------
int main( int argc, char** argv ) {

    int numtasks = 0;
    int task     = 0;
    // INIT ENV
    MPI_( MPI_Init( &argc, &argv ) );
    MPI_( MPI_Errhandler_set( MPI_COMM_WORLD, MPI_ERRORS_RETURN ) );
    MPI_( MPI_Comm_size( MPI_COMM_WORLD, &numtasks ) );
    MPI_( MPI_Comm_rank( MPI_COMM_WORLD, &task  ) );
    std::vector< char > nodeid( MPI_MAX_PROCESSOR_NAME, '\0' );
    int len = 0;
    MPI_( MPI_Get_processor_name( &nodeid[ 0 ], &len ) );

#ifdef MPI_RROBIN_     
    // RETRIEVE TOTAL NUMBER OF NODES USED, is there an easier way ?
    // required to have each GPU assigned to the same number of processes
    // on each node
    const int SEND_NODE_TAG = 0x01;
    //const int SEND_NUM_NODES = 0x10;
    MPI_Request req;
    MPI_( MPI_Isend( &nodeid[ 0 ], MPI_MAX_PROCESSOR_NAME, MPI_CHAR, 0, SEND_NODE_TAG,
                     MPI_COMM_WORLD, &req ) );     
    int node_count = -1;
    if( task == 0 ) {
        typedef std::set< std::string > NodeCount;
        NodeCount ncount;
        std::vector< char > n( MPI_MAX_PROCESSOR_NAME, '\0' );
        MPI_Status s;
        for( int r = 0; r != numtasks; ++r ) {
            MPI_( MPI_Recv( &n[ 0 ], MPI_MAX_PROCESSOR_NAME, MPI_CHAR, r, SEND_NODE_TAG,
                            MPI_COMM_WORLD, &s ) );   
            ncount.insert( &n[ 0 ] );    
        }
        node_count = int( ncount.size() );
#ifndef NO_LOG
        std::cout << "Number of nodes: " << node_count << std::endl;
#endif 
    }
  
    // SEND INFORMATION USED FOR GPU <-> RANK MAPPING TO EACH PROCESS
    // Option 1: use scatter, useful only to send per-process specific information like e.g
    //           the GPU to use. It is in general a more robust method to have the root process
    //           compute the rank -> gpu map
    //std::vector< int > sendbuf( numtasks, node_count );
    // MPI Scatter parameters: address of send buffer,
    //                         per-receiving process receive buffer size,...
    // send buffer size = num tasks x per-receiving-process buffer size
    //MPI_( MPI_Scatter( &sendbuf[ 0 ],  1, MPI_INT, &node_count, 1, MPI_INT, 0, MPI_COMM_WORLD ) ); 
    // Option 2: simply broadcast the number of nodes
    MPI_( MPI_Bcast( &node_count, 1, MPI_INT, 0, MPI_COMM_WORLD ) );
#endif
    // PER TASK DATA INIT - in the real world this is the place where data are read from file
    // through the MPI_File_ functions or, less likely received from the root process
    const int ARRAY_SIZE = 1024 * 1024 * 256;// * 1024 * 256; // 256 Mi floats x 2 == 2 GiB total storage
    // @WARNING: ARRAY_SIZE must be evenly divisible by the number of MPI processes
    const int PER_MPI_TASK_ARRAY_SIZE = ARRAY_SIZE / numtasks;
    if( ARRAY_SIZE % numtasks != 0  && task == 0 ) {
        std::cerr << ARRAY_SIZE << " must be evenly divisible by the number of mpi processes" << std::endl;
        MPI_( MPI_Abort( MPI_COMM_WORLD, 1 ) );
        return 1;
    }
    
    std::vector< real_t > v1( ARRAY_SIZE / numtasks, 0. );
    std::vector< real_t > v2( ARRAY_SIZE / numtasks, 0. );
    for( int i = 0; i != PER_MPI_TASK_ARRAY_SIZE; ++i ) {
        v1[ i ] = 1;
        v2[ i ] = 1;  
    }

    std::vector< double > begins( numtasks );
    std::vector< double > ends( numtasks );
    double begin = clock();
    MPI_( MPI_Gather( &begin, 1, MPI_DOUBLE, &begins[ 0 ], 1, MPI_DOUBLE, 0, MPI_COMM_WORLD ) ); 

    // PARALLEL DOT PRODUCT COMPUTATION
    real_t partial_dot = 0.f;
#ifndef GPU
    int t = 0;
    for( t = 0; t != PER_MPI_TASK_ARRAY_SIZE; ++t ) {
        partial_dot += v1[ t ] * v2[ t ];
    }
    //partial_dot = real_t( p );
#ifndef NO_LOG    
    std::ostringstream os;
    os << &nodeid[ 0 ] << " - rank: " << task << " size: " << PER_MPI_TASK_ARRAY_SIZE 
       << ' ' << t << "  partial dot: " << partial_dot << '\n' ;
    std::cout << os.str(); os.flush();
#endif     
#else
    // SELECT GPU = task % <num gpus on node>, note that with this
    // approach it is possible to support nodes with different numbers of GPUs
    int device_count = 0;
    if( hipGetDeviceCount( &device_count ) != hipSuccess ) {
        std::cerr << task << ' ' << hipGetErrorString( hipGetLastError() ) <<  " hipGetDeviceCount FAILED\n"; 
        MPI_( MPI_Abort( MPI_COMM_WORLD, 1 ) );
        return 1;
    }
#ifdef MPI_RROBIN_
    const int device = ( task / node_count ) % device_count;
#else
    const int device = task % device_count;
#endif
#ifndef NO_LOG
    {    
        std::ostringstream os;
        os << &nodeid[ 0 ] << " - rank: " << task << "\tGPU: " << device << '\n';
        std::cout << os.str(); os.flush();
    }
#endif     
    if( hipSetDevice( device ) != hipSuccess ) {
        std::cerr << task << ' ' << hipGetErrorString( hipGetLastError() ) <<  " cudaGetSetDevice FAILED\n"; 
        MPI_( MPI_Abort( MPI_COMM_WORLD, 1 ) );
        return 1;
    }
#ifdef NO_GPU_MALLOC_TIME
    double malloc_begin = clock();
#endif
    real_t* dev_v1   = 0;
    real_t* dev_v2   = 0;
    real_t* dev_dout = 0;
    if( hipMalloc( &dev_v1,   sizeof( real_t ) * PER_MPI_TASK_ARRAY_SIZE ) != hipSuccess ) {
        std::cerr << task << ' ' << hipGetErrorString( hipGetLastError() ) <<  " hipMalloc FAILED\n"; 
        MPI_( MPI_Abort( MPI_COMM_WORLD, 1 ) );
        return 1;
    }
    if( hipMalloc( &dev_v2,   sizeof( real_t ) * PER_MPI_TASK_ARRAY_SIZE ) != hipSuccess ) {
        std::cerr << task << ' ' << hipGetErrorString( hipGetLastError() ) <<  " hipMalloc FAILED\n"; 
        MPI_( MPI_Abort( MPI_COMM_WORLD, 1 ) );
        return 1;
    }
#ifdef NO_GPU_MALLOC_TIME
    double malloc_end = clock();
    begin += malloc_end - malloc_begin;
#endif
    // MOVE DATA TO GPU
    if( hipMemcpy( dev_v1, &v1[ 0 ], sizeof( real_t ) * PER_MPI_TASK_ARRAY_SIZE,
                    hipMemcpyHostToDevice ) != hipSuccess ) {
        std::cerr << task << ' ' << __LINE__ << ' ' <<  hipGetErrorString( hipGetLastError() ) <<  " hipMemcpy FAILED\n"; 
        MPI_( MPI_Abort( MPI_COMM_WORLD, 1 ) );
        return 1;    
    }
    if( hipMemcpy( dev_v2, &v2[ 0 ], sizeof( real_t ) * PER_MPI_TASK_ARRAY_SIZE,
                    hipMemcpyHostToDevice ) != hipSuccess ) {
        std::cerr << task << ' ' << __LINE__ << ' ' <<  hipGetErrorString( hipGetLastError() ) <<  " hipMemcpy FAILED\n"; 
        MPI_( MPI_Abort( MPI_COMM_WORLD, 1 ) );
        return 1;
    }     
    // INVOKE KERNEL
    const int NUM_THREADS_PER_BLOCK = BLOCK_SIZE; // must match size of buffer used for reduction
    const int NUM_BLOCKS = std::min( PER_MPI_TASK_ARRAY_SIZE  / NUM_THREADS_PER_BLOCK,
                                     0xffff ); // max number of blocks is 64k 
    
#ifndef REDUCE_CPU 
#ifdef NO_GPU_MALLOC_TIME
    malloc_begin = clock();
#endif
    if( hipMalloc( &dev_dout, sizeof( real_t ) * 1 ) != hipSuccess ) {
        std::cerr << task << ' ' << __LINE__ << ' ' << hipGetErrorString( hipGetLastError() ) <<  " hipMalloc FAILED\n"; 
        MPI_( MPI_Abort( MPI_COMM_WORLD, 1 ) );
        return 1;
    }
    // initialize partial dot product to zero
    if( hipMemset( dev_dout, 0, sizeof( real_t) ) != hipSuccess ) {
        std::cerr << task << ' ' << hipGetErrorString( hipGetLastError() ) <<  " hipMemset FAILED\n"; 
        MPI_( MPI_Abort( MPI_COMM_WORLD, 1 ) );
        return 1;
    }
#ifdef NO_GPU_MALLOC_TIME
    malloc_end = clock();
    begin += malloc_end - malloc_begin;
#endif
    // actual on-device computation    
    dot_product_kernel<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>( dev_v1, dev_v2, PER_MPI_TASK_ARRAY_SIZE, dev_dout ); 
    // check for kernel launch errors: it is not possible to catch on-device execution errors but only
    // if there was an error launching the kernel
    if( hipGetLastError() != hipSuccess ) {
        std::cerr << task << ' ' << hipGetErrorString( hipGetLastError() ) <<  " kernel launch FAILED\n"; 
        MPI_( MPI_Abort( MPI_COMM_WORLD, 1 ) );
        return 1;      
    }
    // MOVE DATA TO CPU
    hipMemcpy( &partial_dot, dev_dout, sizeof( real_t ) * 1, hipMemcpyDeviceToHost );
#else
    const int PARTIAL_REDUCE_SIZE = NUM_BLOCKS; 
    if( hipMalloc( &dev_dout, sizeof( real_t ) * PARTIAL_REDUCE_SIZE ) != hipSuccess ) {
        std::cerr << task << ' ' << __LINE__ << ' ' << hipGetErrorString( hipGetLastError() ) <<  " hipMalloc FAILED\n"; 
        MPI_( MPI_Abort( MPI_COMM_WORLD, 1 ) );
        return 1;
    }
    partial_dot_product_kernel<<<NUM_BLOCKS, NUM_THREADS_PER_BLOCK>>>( dev_v1, dev_v2, PER_MPI_TASK_ARRAY_SIZE, dev_dout );  
    std::vector< real_t > rdot( PARTIAL_REDUCE_SIZE );
    hipMemcpy( &rdot[ 0 ], dev_dout, sizeof( real_t ) * PARTIAL_REDUCE_SIZE, hipMemcpyDeviceToHost );
    partial_dot = std::accumulate( rdot.begin(), rdot.end(), 0.f );
#endif

#ifndef NO_LOG
    {    
        std::ostringstream os;
        os << &nodeid[ 0 ] << " - rank: " << task << " partial dot: " << partial_dot << '\n' ;
        std::cout << os.str(); os.flush();
    }
#endif
#endif

    // REDUCE (SUM) ALL ranks -> rank 0
    real_t result = 0.;
    MPI_( MPI_Reduce( &partial_dot, &result, 1, MPI_REAL_T_, MPI_SUM, 0, MPI_COMM_WORLD ) );

    double end = clock();
    MPI_( MPI_Gather( &end, 1, MPI_DOUBLE, &ends[ 0 ], 1, MPI_DOUBLE, 0, MPI_COMM_WORLD ) ); 

    const std::pair< double, double > minmax( *std::min_element( begins.begin(), begins.end() ),
                                              *std::max_element( ends.begin(), ends.end() ) );  


    // IF RANK == 0 -> PRINT RESULT
    if( task == 0 ) {
        std::cout << "dot product result: " << result << std::endl;
        std::cout << "time: " << ( minmax.second - minmax.first ) / CLOCKS_PER_SEC << 's' << std::endl;
    } 
  
#ifdef GPU
    // RELEASE GPU RESOURCES
    hipFree( dev_v1 );
    hipFree( dev_v2 );
    hipFree( dev_dout );
    hipDeviceReset(); 
#endif

    // RELEASE MPI RESOURCES   
    MPI_( MPI_Finalize() );

    return 0;
}
