//Reference implementation of reduction with dot product

//#include <cuda_runtime.h> // automatically added by nvcc

#include <hip/hip_runtime.h>
#include <vector>
#include <iostream>


typedef float real_t;

const size_t BLOCK_SIZE = 16;

__global__ void full_dot( const real_t* v1, const real_t* v2, real_t* out, int N ) {
    __shared__ real_t cache[ BLOCK_SIZE ];
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    cache[ threadIdx.x ] = 0.f;
    while( i < N ) {
        cache[ threadIdx.x ] += v1[ i ] * v2[ i ];
        i += gridDim.x * blockDim.x;
    }    
    i = BLOCK_SIZE / 2;
    while( i > 0 ) {
        if( threadIdx.x < i ) cache[ threadIdx.x ] += cache[ threadIdx.x + i ];
        __syncthreads();
        i /= 2; //not sure bitwise operations are actually faster
    }
#ifndef NO_SYNC // serialized access to shared data; 
    if( threadIdx.x == 0 ) atomicAdd( out, cache[ 0 ] );
#else // no sync, what most likely happens is:
      // 1) all threads read 0
      // 2) all threads write concurrently 16 (local block dot product)
    if( threadIdx.x == 0 ) *out += cache[ 0 ];
#endif                
    
}

real_t dot( const real_t* v1, const real_t* v2, int N ) {
    real_t s = 0;
    for( int i = 0; i != N; ++i ) {
        s += v1[ i ] * v2[ i ];
    }
    return s;
}


__global__ void init_vector( real_t* v, int N ) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    while( i < N ) {
        v[ i ] = 1.0f;//real_t( i ) / 1000000.f;
        i += gridDim.x * blockDim.x;
    } 
}


//------------------------------------------------------------------------------
int main(int argc, char** argv ) {
    
    const size_t ARRAY_SIZE = 1024;//1024 * 1024; //1Mi elements
    const int BLOCKS = 64;//512;
    const int THREADS_PER_BLOCK = BLOCK_SIZE;//256; // total threads = 512 x 256 = 128ki threads;                                   
    const size_t SIZE = ARRAY_SIZE * sizeof( real_t );
    
    // device storage
    real_t* dev_v1 = 0;  // vector 1
    real_t* dev_v2 = 0;  // vector 2
    real_t* dev_out = 0; // result
    hipMalloc( &dev_v1,  SIZE );
    hipMalloc( &dev_v2,  SIZE );
    hipMalloc( &dev_out, sizeof( real_t ) );

    // host storage
    std::vector< real_t > host_v1( ARRAY_SIZE );
    std::vector< real_t > host_v2( ARRAY_SIZE );
    real_t host_out = 0.f;

    // initialize vector 1 with kernel; much faster than using for loops on the cpu
    init_vector<<< 1024, 256  >>>( dev_v1, ARRAY_SIZE );
    hipMemcpy( &host_v1[ 0 ], dev_v1, SIZE, hipMemcpyDeviceToHost );
    // initialize vector 2 with kernel; much faster than using for loops on the cpu
    init_vector<<< 1024, 256  >>>( dev_v2, ARRAY_SIZE );
    hipMemcpy( &host_v2[ 0 ], dev_v2, SIZE, hipMemcpyDeviceToHost );
    
    // initialize result on GPU: note the use of cudaMemset, alternatives are to run a kernel
    // or copy from CPU
    hipMemset( dev_out, 0, sizeof( real_t) );   

    // execute kernel
    full_dot<<<BLOCKS, THREADS_PER_BLOCK>>>( dev_v1, dev_v2, dev_out, ARRAY_SIZE );
    std::cout << hipGetErrorString( hipGetLastError() ) << std::endl;
             
    // copy output data from device(gpu) to host(cpu)
    hipMemcpy( &host_out, dev_out, sizeof( real_t ), hipMemcpyDeviceToHost );

    // print dot product by summing up the partially reduced vectors
    std::cout << "GPU: " << host_out << std::endl;    

    // print dot product on cpu
    std::cout << "CPU: " << dot( &host_v1[ 0 ], &host_v2[ 0 ], ARRAY_SIZE ) << std::endl;

    // free memory
    hipFree( dev_v1 );
    hipFree( dev_v2 );
    hipFree( dev_out );

    return 0;
}
